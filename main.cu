#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <iomanip>
#include <vector>
#include <math.h>

#include <chrono>

#define DEBUG 1

#define NULL_ADDRESS_MARGIN 4
#define NULL_ADDRESS_READ (0x10000 + 0)
#define NULL_ADDRESS_WRITE (0x10000 + 2)
#define STACK_ZERO 0x0100

typedef const uint8_t flag_t;
typedef const uint16_t flag16_t;
typedef const uint8_t int_t;
typedef const int8_t int_signed_t;
typedef uint16_t int16u_t;
typedef uint8_t int8u_t;
typedef uint8_t bit_t;

struct SystemState;
struct ComputationState;

__device__
void operationTransition(uint8_t, SystemState*, ComputationState*);

#include "states.h"
#include "regions.h"
#include "_instructions.h"

__device__
void operationTransition(uint8_t opcode, SystemState* state, ComputationState* computation_state) {
    instructions[opcode].transition(state, computation_state);
}

/* */

__global__
void add(int num_states, uint32_t num_instructions, SystemState *states)
{
    for (int i = 0; i < num_instructions; i++) {
        states[threadIdx.x].next();
    }
}

#include "utilities.h"

/* */

int main(void)
{
    // int num_states = 256;
    int num_states = 15;
    SystemState *states;

    uint64_t num_instructions = 0;
    std::cin >> num_instructions;

    std::cout << "NUM_INSTRUCTIONS [ " << num_instructions << ", " << sizeof(SystemState) << " ]\n\n";

    hipMallocManaged(&states, num_states*sizeof(SystemState));

    /* */

    std::vector<char> file_data = fileRead("data/nestest.program");
    std::vector<std::vector<std::string>> log_lines = logRead("data/nestest.log");

    for (int i = 0; i < num_states; i++) {
        states[i] = SystemState(file_data, 0xC000 + i - 7, 0xC000);
    }

    /* */

    auto start = std::chrono::high_resolution_clock::now();

    add<<<1, num_states>>>(num_states, num_instructions, states);
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();

    std::cout << "\n> " << std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count() << "\n\n";

    /* */

    #ifdef DEBUG

    std::cout << "\n";
    for (int i = 0; i < num_states; i++) {
        if (i == 7 || i == 8) {
            std::cout << "--------------------------" << "\n";
        }

        std::cout << traceLineFormat(states[i].traceLineData[states[i].traceIndex-1]) << "\n";
    }

    std::cout << "\n";

    /* */

    int mismatch_count = 0;
    for (int i = 0; i < states[7].traceIndex; i++) {
        std::string reference = logLineFormat(log_lines[i]);
        std::string actual = traceLineFormat(states[7].traceLineData[i]);

        if (reference == actual) {
            std::cout << std::hex << std::setw(2) << std::setfill('0') << std::uppercase << i << " ";
            std::cout << "│ " << actual << "\n";
        } else {
            std::cout << "\n" << std::hex << std::setw(2) << std::setfill('0') << std::uppercase << i << "";
            std::cout << " · " << reference << "\n";
            std::cout << "     " << lineCompare(reference, actual) << "\n";

            mismatch_count++;
        }
    }

    std::cout << "\n" << mismatch_count << "\n" << std::endl;

    #endif

    /* */

    hipFree(states);

    return 0;
}